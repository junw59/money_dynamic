#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <random>
#include <iostream>
#include <fstream>
#include <math.h>
#include <string>

using namespace std;

#define SIGMA 0.7071

#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#if defined(_WIN32)
#include <chrono>
int gettimeofday(struct timeval* tp, struct timezone* tzp) {
    namespace sc = std::chrono;
    sc::system_clock::duration d = sc::system_clock::now().time_since_epoch();
    sc::seconds s = sc::duration_cast<sc::seconds>(d);
    tp->tv_sec = s.count();
    tp->tv_usec = sc::duration_cast<sc::microseconds>(d - s).count();

    return 0;
}
#endif // _WIN32


double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}

#define CHECK(call)\
{\
    const hipError_t error=call;\
    if(error!=hipSuccess)\
    {\
        printf("ERROR: %s:%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
        exit(1);\
    }\
}

int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;

    // HANDLE_ERROR(cudaGetDeviceCount(&count));
    hipGetDeviceCount(&count);
    printf("gpu num %d\n", count);
    // HANDLE_ERROR(cudaGetDeviceProperties(&prop, 0));
    hipGetDeviceProperties(&prop, 0);
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

void initDevice(int devNum)
{
    int dev = devNum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using device %d: %s\n",dev,deviceProp.name);
    CHECK(hipSetDevice(dev));
}

__global__ void singlerealization(float *agents, int size, int repeat_times, float time_step, float* rds ){
    // int realiza = threadIdx+blockDim*blockIdx;
    int realiza = threadIdx.x+blockDim.x*blockIdx.x;
    if ( realiza < repeat_times ){
        // if (realiza%1024 == 0) printf("!!%d\n",realiza);
        // random_device rd; //获取随机数种子
        // mt19937 gen(rd() + time(0) + rds); //Standard mersenne_twister_engine seeded with rd()
        // normal_distribution<double> nd(1,1);
        float bar_v = 0;
        for (int i = 0; i < size; i++){
            bar_v += agents[i+size*realiza];
        }
        bar_v = bar_v/(size - 1);
        // cout << bar_v << endl;

        for (int i = 0; i < size; i++){
            float factor = 1 - size * time_step / (size - 1) + SIGMA * rds[i+size*realiza] * sqrtf(2 * time_step);
            agents[i+size*realiza] = agents[i+size*realiza] * factor + bar_v * time_step;
            // agents[i+size*realiza] = agents[i+size*realiza] + 1;
            // float factor = 1 + SIGMA * rds[i+size*realiza] * sqrtf(2 * time_step);
            // agents[i+size*realiza] = agents[i+size*realiza] * factor;
            // agents[i+size*realiza] = rds[i+size*realiza];
            // agents[i+size*realiza] = i+size*realiza + rds[i+size*realiza];
        }
    }
}

__global__ void cal_vi_m(float *agents, int size, int repeat_times, float* vims_dev ){
    // int realiza = threadIdx+blockDim*blockIdx;
    int realiza = threadIdx.x+blockDim.x*blockIdx.x;
    if ( realiza < repeat_times ){
        float vim = 0;
        for (int i = 0; i < size; i++){
            vim += agents[i+size*realiza];
        }
        vims_dev[realiza] = vim / size;
    }
}

__global__ void cal_vi2(float *agents, int size, int repeat_times, float* vi2s_dev ){
    // int realiza = threadIdx+blockDim*blockIdx;
    int realiza = threadIdx.x+blockDim.x*blockIdx.x;
    if ( realiza < repeat_times ){
        float vim = 0;
        for (int i = 0; i < size; i++){
            vim += agents[i+size*realiza] * agents[i+size*realiza];
        }
        vi2s_dev[realiza] = vim / size;
    }
}

__global__ void cal_vivj(float *agents, int size, int repeat_times, float* vivjs_dev ){
    // int realiza = threadIdx+blockDim*blockIdx;
    int realiza = threadIdx.x+blockDim.x*blockIdx.x;
    if ( realiza < repeat_times ){
        double vivj = 0;
        int num = 0;
        for (int i = 0; i < size; i++){
            for (int j = i + 1; j < size; j++){
                // vivj += agents[i+size*realiza] * agents[j+size*realiza];
                num ++;
                // vivj = agents[i+size*realiza] * agents[j+size*realiza] / num + vivj/num+(num-1);
                vivj = vivj + (agents[i+size*realiza] * agents[j+size*realiza] - vivj) / num;
            }
        }
        vivjs_dev[realiza] = vivj;
    }
}

float cal_mean(float* numbes, int size){
    float m = 0;
    for (int i = 0; i < size; i++){
        m += numbes[i];
    }
    return m / size;
}

void save_snap(string path, float *agents, int size, float repeat_times){
    ofstream write_f(path);
    for (int t = 0; t < repeat_times; t++){
        for (int i = 0; i < size; i++){
            write_f << agents[i+t*size] << ",";
        }
        write_f << endl;
    }
}

void singlestep(float *agents, int size, float time_step, int rds){
    random_device rd; //获取随机数种子
    mt19937 gen(rd() + time(0) + rds); //Standard mersenne_twister_engine seeded with rd()
    normal_distribution<double> nd(0,1);
    float bar_v = 0;
    for (int i = 0; i < size; i++){
        bar_v += agents[i];
    }
    bar_v = bar_v/(size - 1);
    // cout << bar_v << endl;

    for (int i = 0; i < size; i++){
        // float factor = 1 - size * time_step / (size - 1) + SIGMA * nd(gen) * sqrtf(2 * time_step);
        // agents[i] = agents[i] * factor + bar_v * time_step;
        float factor = 0 + SIGMA * nd(gen) * sqrtf(2 * time_step);
        agents[i] = agents[i] * factor;
    }

    return ;
}

int main(){
    initDevice(0);

    random_device rd; //获取随机数种子
    mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
    // uniform_real_distribution<float> distr(-1, 1);
    normal_distribution<float> nd(0,1);
    string savepath = "./cuda_data_time001.txt";
    ofstream write_f(savepath);
    // for (int i = 0; i < 5; i++){
    //     cout << nd(gen) << "\t";
    // }
    // getThreadNum();
    const int size_agents = 10000;
    const float delta_T = 0.01;
    const int Times = 1000/delta_T;
    const int repeat_times = 10240;
    cout << repeat_times << endl;

    int nBytes = sizeof(float)*repeat_times;
    float* agents_rep_host = (float*)malloc(nBytes*size_agents);
    float* agents_rep_from_dev = (float*)malloc(nBytes*size_agents);
    float* noise_rds_host = (float*)malloc(nBytes*size_agents);
    float* vims_host = (float*)malloc(nBytes);
    float* vi2s_host = (float*)malloc(nBytes);
    float* vivjs_host = (float*)malloc(nBytes);
    for (int i = 0; i < repeat_times*size_agents; i++) agents_rep_host[i] = 1;

    // ofstream data("noise.txt");
    // for (int i = 0; i < repeat_times*size_agents; i++) data << noise_rds_host[i] << ",";


    // return 0;
    // cudaMalloc, 开辟 device 内存
    float* agents_rep_dev = NULL ;
    float* noise_rds_dev = NULL ;
    float* vims_dev = NULL ;
    float* vi2s_dev = NULL ;
    float* vivjs_dev = NULL ;

    CHECK(hipMalloc((void**)&agents_rep_dev, nBytes*size_agents)); 
    CHECK(hipMalloc((void**)&noise_rds_dev, nBytes*size_agents)); 
    CHECK(hipMalloc((void**)&vims_dev, nBytes));
    CHECK(hipMalloc((void**)&vi2s_dev, nBytes));
    CHECK(hipMalloc((void**)&vivjs_dev, nBytes));

    //输入数据从主机内存拷贝到设备内存
    CHECK(hipMemcpy(agents_rep_dev, agents_rep_host, nBytes*size_agents, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(noise_rds_dev, noise_rds_host, nBytes*size_agents, hipMemcpyHostToDevice));
    // CHECK(cudaMemcpy(vims_dev, vims_host, nBytes, cudaMemcpyHostToDevice));
    // CHECK(cudaMemcpy(vi2s_dev, vi2s_host, nBytes, cudaMemcpyHostToDevice));
    // CHECK(cudaMemcpy(vivjs_dev, vivjs_host, nBytes, cudaMemcpyHostToDevice));

    string a = "./cuda_data_time001.-1.txt";
    save_snap(a, agents_rep_host, size_agents, repeat_times);

    // //一维线程块，32×32
    dim3 block(1024);
    // //一维线程网格
    dim3 grid((repeat_times-1)/block.x+1);
    cout << block.x << "\t" << grid.x << endl;
    double gpuStart = cpuSecond();
    cout << "start gpu" << endl;
    for (int t = 0; t < Times; t++) {
        for (int i = 0; i < repeat_times*size_agents; i++){
            noise_rds_host[i] = nd(gen);
        }
        CHECK(hipMemcpy(noise_rds_dev, noise_rds_host, nBytes*size_agents, hipMemcpyHostToDevice));

        singlerealization<<<grid, block >>>( agents_rep_dev, size_agents, repeat_times, delta_T, noise_rds_dev);
        cal_vi_m<<<grid, block >>>( agents_rep_dev, size_agents, repeat_times, vims_dev);
        cal_vi2<<<grid, block >>>( agents_rep_dev, size_agents, repeat_times, vi2s_dev);
        cal_vivj<<<grid, block >>>( agents_rep_dev, size_agents, repeat_times, vivjs_dev);
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(vims_host, vims_dev, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(vi2s_host, vi2s_dev, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(vivjs_host, vivjs_dev, nBytes, hipMemcpyDeviceToHost));
        float vim1;
        float vim2;
        float vim3;
        vim1 = cal_mean(vims_host, repeat_times);
        vim2 = cal_mean(vi2s_host, repeat_times);
        vim3 = cal_mean(vivjs_host, repeat_times);
        if (t%100 == 0) cout << t << "\t";
        // write_f << t << "," << vim1 << "," << endl;
        write_f << t << "," << vim1 << "," << vim2 << "," << vim3 << endl;

        if (t%1000 == 0){
            CHECK(hipMemcpy(agents_rep_host, agents_rep_dev, nBytes*size_agents, hipMemcpyDeviceToHost));
            string a = "./cuda_data_time001."+to_string(t)+".txt";
            save_snap(a, agents_rep_host, size_agents, repeat_times);
        }
    }

    double gpuTime = cpuSecond() - gpuStart;
    printf("GPU Execution Time: %f sec\n", (double) gpuTime);


    double cpuStart=cpuSecond();
    for ( int realiza = 0; realiza < repeat_times; realiza++){ // 对不同的 realization 计算
        // if ( int(realiza%1000) == 0) cout << realiza << "\t";
        singlestep( agents_rep_host+size_agents*realiza, size_agents, delta_T, rd());
    }
    double cpuTime = cpuSecond() - cpuStart;
    printf("CPU Execution Time: %f sec\n", (double) cpuTime);

    CHECK(hipMemcpy(agents_rep_from_dev, agents_rep_dev, nBytes*size_agents, hipMemcpyDeviceToHost));
    // CHECK(cudaMemcpy(vims_host, vims_dev, nBytes, cudaMemcpyDeviceToHost));
    // CHECK(cudaMemcpy(vi2s_host, vi2s_dev, nBytes, cudaMemcpyDeviceToHost));
    // CHECK(cudaMemcpy(vivjs_host, vivjs_dev, nBytes, cudaMemcpyDeviceToHost));

    // for (int i = 0; i < 10; i++){
    //     cout << agents_rep_from_dev[i] << "\t" << agents_rep_host[i] << "\t";
    // }

    hipFree(agents_rep_dev);
    free(agents_rep_from_dev);
    free(agents_rep_host);
    hipDeviceReset();

    return 0;
}